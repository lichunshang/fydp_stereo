#include "hip/hip_runtime.h"
/**
    This file is part of sgm. (https://github.com/dhernandez0/sgm).

    Copyright (c) 2016 Daniel Hernandez Juarez.

    sgm is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    sgm is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with sgm.  If not, see <http://www.gnu.org/licenses/>.

**/

#include "do_disp.h"

bool directory_exists(const char* dir) {
	DIR* d = opendir(dir);
	bool ok = false;
	if(d) {
	    closedir(d);
	    ok = true;
	}
	return ok;
}

void disparity_errors(cv::Mat estimation, const char* gt_file, int *n, int *n_err) {
	int nlocal = 0;
	int nerrlocal = 0;

	cv::Mat gt_image = cv::imread(gt_file, cv::IMREAD_UNCHANGED);
	if(!gt_image.data) {
		std::cerr << "Couldn't read the file " << gt_file << std::endl;
		exit(EXIT_FAILURE);
	}
	if(estimation.rows != gt_image.rows || estimation.cols != gt_image.cols) {
		std::cerr << "Ground truth must have the same dimesions" << std::endl;
		exit(EXIT_FAILURE);
	}
	const int type = estimation.type();
	const uchar depth = type & CV_MAT_DEPTH_MASK;
	for(int i = 0; i < gt_image.rows; i++) {
		for(int j = 0; j < gt_image.cols; j++) {
			const uint16_t gt = gt_image.at<uint16_t>(i, j);
			if(gt > 0) {
				const float gt_f = ((float)gt)/256.0f;
				float est;
				if(depth == CV_8U) {
					est = (float) estimation.at<uint8_t>(i, j);
				} else {
					est = estimation.at<float>(i, j);
				}
				const float err = fabsf(est-gt_f);
				const float ratio = err/fabsf(gt_f);
				if(err > ABS_THRESH && ratio > REL_THRESH) {
					nerrlocal++;
				}
				nlocal++;
			}
		}
	}
	*n += nlocal;
	*n_err += nerrlocal;
}

bool check_directories_exist(const char* directory, const char* left_dir, const char* right_dir, const char* disparity_dir) {
	char left_dir_sub[PATH_MAX];
	char right_dir_sub[PATH_MAX];
	char disparity_dir_sub[PATH_MAX];
	sprintf(left_dir_sub, "%s/%s", directory, left_dir);
	sprintf(right_dir_sub, "%s/%s", directory, right_dir);
	sprintf(disparity_dir_sub, "%s/%s", directory, disparity_dir);

	return directory_exists(left_dir_sub) && directory_exists(right_dir_sub) && directory_exists(disparity_dir_sub);
}

cv::Mat do_disp(int param1, int param2, cv::Mat lf, cv::Mat rf) {

	std::vector<float> times;

	init_disparity_method(param1, param2);
	cv::Mat disparity_im;

	cv::Mat h_im0 = lf;
	if(!h_im0.data) {
		std::cerr << "Couldn't read the file " << std::endl;
	}
	cv::Mat h_im1 = rf;
	if(!h_im1.data) {
		std::cerr << "Couldn't read the file " << std::endl;
	}

	// Convert images to grayscale
	if (h_im0.channels()>1) {
		cv::cvtColor(h_im0, h_im0, CV_RGB2GRAY);
	}

	if (h_im1.channels()>1) {
		cv::cvtColor(h_im1, h_im1, CV_RGB2GRAY);
	}

	if(h_im0.rows != h_im1.rows || h_im0.cols != h_im1.cols) {
		std::cerr << "Both images must have the same dimensions" << std::endl;
	}

#if LOG
	std::cout << "processing: " << std::endl;
#endif
	// Compute
	float elapsed_time_ms;
	disparity_im = compute_disparity_method(h_im0, h_im1, &elapsed_time_ms);
#if LOG
	std::cout << "done" << std::endl;
#endif
	times.push_back(elapsed_time_ms);

	finish_disparity_method();

	double mean = std::accumulate(times.begin(), times.end(), 0.0) / times.size();

	std::cout << "It took an average of " << mean << " miliseconds, " << 1000.0f/mean << " fps" << std::endl;

	return disparity_im;
}
